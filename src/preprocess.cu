#include "preprocess.h"

void copy_lpinfo_to_device(const LP_info_cpu *lp_info_cpu, LP_info_gpu *lp_info_gpu) {
    int m = lp_info_cpu->m;
    int n = lp_info_cpu->n;

    lp_info_gpu->m = m;
    lp_info_gpu->n = n;
    lp_info_gpu->obj_constant = lp_info_cpu->obj_constant;

    // Copy A to GPU
    lp_info_gpu->A = new sparseMatrix;
    transfer_CSR_matrix(lp_info_cpu->A, lp_info_gpu->A);
    
    // Generate AT on CPU first, then transfer to GPU
    lp_info_gpu->AT = new sparseMatrix;
    sparseMatrix AT_host;
    CSR_transpose_host(*(lp_info_cpu->A), &AT_host);
    transfer_CSR_matrix(&AT_host, lp_info_gpu->AT);
    
    // Free the temporary host AT
    free(AT_host.value);
    free(AT_host.colIndex);
    free(AT_host.rowPtr);

    CUDA_CHECK(hipMalloc(&lp_info_gpu->AL, m * sizeof(HPRLP_FLOAT)));
    CUDA_CHECK(hipMemcpy(lp_info_gpu->AL, lp_info_cpu->AL, m * sizeof(HPRLP_FLOAT), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMalloc(&lp_info_gpu->AU, m * sizeof(HPRLP_FLOAT)));
    CUDA_CHECK(hipMemcpy(lp_info_gpu->AU, lp_info_cpu->AU, m * sizeof(HPRLP_FLOAT), hipMemcpyHostToDevice));

    CUDA_CHECK(hipMalloc(&lp_info_gpu->l, n * sizeof(HPRLP_FLOAT)));
    CUDA_CHECK(hipMemcpy(lp_info_gpu->l, lp_info_cpu->l, n * sizeof(HPRLP_FLOAT), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMalloc(&lp_info_gpu->u, n * sizeof(HPRLP_FLOAT)));
    CUDA_CHECK(hipMemcpy(lp_info_gpu->u, lp_info_cpu->u, n * sizeof(HPRLP_FLOAT), hipMemcpyHostToDevice));

    CUDA_CHECK(hipMalloc(&lp_info_gpu->c, n * sizeof(HPRLP_FLOAT)));
    CUDA_CHECK(hipMemcpy(lp_info_gpu->c, lp_info_cpu->c, n * sizeof(HPRLP_FLOAT), hipMemcpyHostToDevice));
}


void prepare_spmv(HPRLP_workspace_gpu *workspace) {
    int n = workspace->n;
    int m = workspace->m;
    workspace->spmv_A = new CUSPARSE_spmv_A;
    workspace->spmv_AT = new CUSPARSE_spmv_AT;
    hipsparseHandle_t cusparseHandle;
    hipsparseCreate(&cusparseHandle);
    workspace->spmv_A->cusparseHandle = cusparseHandle;
    workspace->spmv_AT->cusparseHandle = cusparseHandle;
    workspace->spmv_A->alpha = 1.0;
    workspace->spmv_A->beta = 0.0;
    workspace->spmv_AT->alpha = 1.0;
    workspace->spmv_AT->beta = 0.0;
    workspace->spmv_A->_operator = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    workspace->spmv_A->computeType = HIP_R_64F;
    workspace->spmv_A->alg = HIPSPARSE_SPMV_CSR_ALG2;
    workspace->spmv_AT->_operator = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    workspace->spmv_AT->computeType = HIP_R_64F;
    workspace->spmv_AT->alg = HIPSPARSE_SPMV_CSR_ALG2;
    hipsparseCreateDnVec(&workspace->spmv_A->x_bar_cusparseDescr, n, workspace->x_bar, HIP_R_64F);
    hipsparseCreateDnVec(&workspace->spmv_A->x_hat_cusparseDescr, n, workspace->x_hat, HIP_R_64F);
    hipsparseCreateDnVec(&workspace->spmv_A->x_temp_cusparseDescr, n, workspace->x_temp, HIP_R_64F);
    hipsparseCreateDnVec(&workspace->spmv_AT->y_bar_cusparseDescr, m, workspace->y_bar, HIP_R_64F);
    hipsparseCreateDnVec(&workspace->spmv_AT->y_cusparseDescr, m, workspace->y, HIP_R_64F);
    hipsparseCreateDnVec(&workspace->spmv_AT->ATy_cusparseDescr, n, workspace->ATy, HIP_R_64F);
    hipsparseCreateDnVec(&workspace->spmv_A->Ax_cusparseDescr, m, workspace->Ax, HIP_R_64F);

    // CSR Sparse Matrix Descriptor
    hipsparseCreateCsr(&workspace->spmv_A->A_cusparseDescr, workspace->m, workspace->n, workspace->A->numElements,
                workspace->A->rowPtr, workspace->A->colIndex, workspace->A->value,
                HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);

    hipsparseCreateCsr(&workspace->spmv_AT->AT_cusparseDescr, workspace->n, workspace->m, workspace->AT->numElements,
                workspace->AT->rowPtr, workspace->AT->colIndex, workspace->AT->value,
                HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);

    
    hipsparseSpMV_bufferSize(cusparseHandle,workspace->spmv_A->_operator,
                            &workspace->spmv_A->alpha, workspace->spmv_A->A_cusparseDescr, workspace->spmv_A->x_bar_cusparseDescr,
                            &workspace->spmv_A->beta, workspace->spmv_A->Ax_cusparseDescr, workspace->spmv_A->computeType,
                            workspace->spmv_A->alg, &workspace->spmv_A->buffersize);

    hipMalloc(&workspace->spmv_A->buffer, workspace->spmv_A->buffersize);

    hipsparseSpMV_preprocess(cusparseHandle,workspace->spmv_A->_operator,
                            &workspace->spmv_A->alpha, workspace->spmv_A->A_cusparseDescr, workspace->spmv_A->x_bar_cusparseDescr,
                            &workspace->spmv_A->beta, workspace->spmv_A->Ax_cusparseDescr, workspace->spmv_A->computeType,
                            workspace->spmv_A->alg, workspace->spmv_A->buffer);

    hipsparseSpMV(cusparseHandle,workspace->spmv_A->_operator,
                        &workspace->spmv_A->alpha, workspace->spmv_A->A_cusparseDescr, workspace->spmv_A->x_bar_cusparseDescr,
                        &workspace->spmv_A->beta, workspace->spmv_A->Ax_cusparseDescr, workspace->spmv_A->computeType,
                        workspace->spmv_A->alg, workspace->spmv_A->buffer);

    hipsparseSpMV_bufferSize(cusparseHandle,workspace->spmv_AT->_operator,
                        &workspace->spmv_AT->alpha, workspace->spmv_AT->AT_cusparseDescr, workspace->spmv_AT->y_bar_cusparseDescr,
                        &workspace->spmv_AT->beta, workspace->spmv_AT->ATy_cusparseDescr, workspace->spmv_AT->computeType,
                        workspace->spmv_AT->alg, &workspace->spmv_AT->buffersize);

    hipMalloc(&workspace->spmv_AT->buffer, workspace->spmv_AT->buffersize);

    hipsparseSpMV_preprocess(cusparseHandle,workspace->spmv_AT->_operator,
                        &workspace->spmv_AT->alpha, workspace->spmv_AT->AT_cusparseDescr, workspace->spmv_AT->y_bar_cusparseDescr,
                        &workspace->spmv_AT->beta, workspace->spmv_AT->ATy_cusparseDescr, workspace->spmv_AT->computeType,
                        workspace->spmv_AT->alg, workspace->spmv_AT->buffer);
}


void allocate_memory(HPRLP_workspace_gpu *workspace, LP_info_gpu *lp_info_gpu) {
    // allocate memory for the workspace
    int m = workspace->m;
    int n = workspace->n;


    create_zero_vector_device(workspace->x, n);
    create_zero_vector_device(workspace->last_x, n);
    create_zero_vector_device(workspace->x_temp, n);
    create_zero_vector_device(workspace->x_hat, n);
    create_zero_vector_device(workspace->x_bar, n);
    create_zero_vector_device(workspace->y, m);
    create_zero_vector_device(workspace->last_y, m);
    create_zero_vector_device(workspace->y_temp, m);
    create_zero_vector_device(workspace->y_bar, m);
    create_zero_vector_device(workspace->y_hat, m);
    create_zero_vector_device(workspace->y_obj, m);
    create_zero_vector_device(workspace->z_bar, n);

    workspace->A = lp_info_gpu->A;
    workspace->AT = lp_info_gpu->AT;
    workspace->AL = lp_info_gpu->AL;
    workspace->AU = lp_info_gpu->AU;
    workspace->c = lp_info_gpu->c;
    workspace->l = lp_info_gpu->l;
    workspace->u = lp_info_gpu->u;

    create_zero_vector_device(workspace->Rd, n);
    create_zero_vector_device(workspace->Rp, m);
    create_zero_vector_device(workspace->ATy, n);
    create_zero_vector_device(workspace->Ax, m);

    workspace->check = false;

    hipblasCreate(&workspace->cublasHandle);
    
    prepare_spmv(workspace);
}


void free_workspace(HPRLP_workspace_gpu *workspace) {
    /*
     * Free all GPU memory allocated in allocate_memory and prepare_spmv.
     * This prevents memory leaks. Note: When called from Python ctypes, the
     * process may still segfault during Python interpreter shutdown due to
     * CUDA/ctypes interaction, but this is harmless (happens after results returned).
     */
    if (!workspace) return;
    
    // Destroy cuBLAS handle FIRST (before freeing vectors it might reference)
    if (workspace->cublasHandle) {
        hipblasDestroy(workspace->cublasHandle);
        workspace->cublasHandle = nullptr;
    }
    
    // Destroy CUSPARSE descriptors BEFORE freeing the underlying memory
    // Free CUSPARSE resources for AT matrix operations
    if (workspace->spmv_AT) {
        if (workspace->spmv_AT->y_bar_cusparseDescr) hipsparseDestroyDnVec(workspace->spmv_AT->y_bar_cusparseDescr);
        if (workspace->spmv_AT->y_cusparseDescr) hipsparseDestroyDnVec(workspace->spmv_AT->y_cusparseDescr);
        if (workspace->spmv_AT->ATy_cusparseDescr) hipsparseDestroyDnVec(workspace->spmv_AT->ATy_cusparseDescr);
        if (workspace->spmv_AT->AT_cusparseDescr) hipsparseDestroySpMat(workspace->spmv_AT->AT_cusparseDescr);
        if (workspace->spmv_AT->buffer) hipFree(workspace->spmv_AT->buffer);
        // Destroy shared cusparse handle (only once, shared between spmv_A and spmv_AT)
        if (workspace->spmv_AT->cusparseHandle) {
            hipsparseDestroy(workspace->spmv_AT->cusparseHandle);
        }
        delete workspace->spmv_AT;
        workspace->spmv_AT = nullptr;
    }
    
    // Free CUSPARSE resources for A matrix operations
    if (workspace->spmv_A) {
        if (workspace->spmv_A->x_bar_cusparseDescr) hipsparseDestroyDnVec(workspace->spmv_A->x_bar_cusparseDescr);
        if (workspace->spmv_A->x_hat_cusparseDescr) hipsparseDestroyDnVec(workspace->spmv_A->x_hat_cusparseDescr);
        if (workspace->spmv_A->x_temp_cusparseDescr) hipsparseDestroyDnVec(workspace->spmv_A->x_temp_cusparseDescr);
        if (workspace->spmv_A->Ax_cusparseDescr) hipsparseDestroyDnVec(workspace->spmv_A->Ax_cusparseDescr);
        if (workspace->spmv_A->A_cusparseDescr) hipsparseDestroySpMat(workspace->spmv_A->A_cusparseDescr);
        if (workspace->spmv_A->buffer) hipFree(workspace->spmv_A->buffer);
        // Note: cusparseHandle already destroyed above with spmv_AT
        delete workspace->spmv_A;
        workspace->spmv_A = nullptr;
    }
    
    // NOW free device vectors (after descriptors are destroyed)
    if (workspace->x) hipFree(workspace->x);
    if (workspace->last_x) hipFree(workspace->last_x);
    if (workspace->x_temp) hipFree(workspace->x_temp);
    if (workspace->x_hat) hipFree(workspace->x_hat);
    if (workspace->x_bar) hipFree(workspace->x_bar);
    if (workspace->y) hipFree(workspace->y);
    if (workspace->last_y) hipFree(workspace->last_y);
    if (workspace->y_temp) hipFree(workspace->y_temp);
    if (workspace->y_bar) hipFree(workspace->y_bar);
    if (workspace->y_hat) hipFree(workspace->y_hat);
    if (workspace->y_obj) hipFree(workspace->y_obj);
    if (workspace->z_bar) hipFree(workspace->z_bar);
    if (workspace->Rd) hipFree(workspace->Rd);
    if (workspace->Rp) hipFree(workspace->Rp);
    if (workspace->ATy) hipFree(workspace->ATy);
    if (workspace->Ax) hipFree(workspace->Ax);
    
    // Note: A, AT, AL, AU, c, l, u are just pointers to lp_info_gpu data.
    // They should NOT be freed here - they will be freed in free_lp_info().
}


void free_lp_info(LP_info_gpu *lp_info) {
    /*
     * Free GPU memory allocated in copy_lpinfo_to_device.
     */
    if (!lp_info) return;
    
    // Free sparse matrices A and AT
    if (lp_info->A) {
        if (lp_info->A->rowPtr) hipFree(lp_info->A->rowPtr);
        if (lp_info->A->colIndex) hipFree(lp_info->A->colIndex);
        if (lp_info->A->value) hipFree(lp_info->A->value);
        delete lp_info->A;
    }
    
    if (lp_info->AT) {
        if (lp_info->AT->rowPtr) hipFree(lp_info->AT->rowPtr);
        if (lp_info->AT->colIndex) hipFree(lp_info->AT->colIndex);
        if (lp_info->AT->value) hipFree(lp_info->AT->value);
        delete lp_info->AT;
    }
    
    // Free constraint and variable bound vectors
    if (lp_info->AL) hipFree(lp_info->AL);
    if (lp_info->AU) hipFree(lp_info->AU);
    if (lp_info->l) hipFree(lp_info->l);
    if (lp_info->u) hipFree(lp_info->u);
    if (lp_info->c) hipFree(lp_info->c);
}

void free_lp_info_cpu(LP_info_cpu *lp_info) {
    /*
     * Free CPU memory allocated in build_model_from_arrays() or build_model_from_mps().
     * This is used to clean up LP_info_cpu structures after solving.
     * Note: AT is no longer stored in LP_info_cpu; it's generated on-the-fly
     */
    if (!lp_info) return;
    
    // Free sparse matrix A
    if (lp_info->A) {
        if (lp_info->A->rowPtr) free(lp_info->A->rowPtr);
        if (lp_info->A->colIndex) free(lp_info->A->colIndex);
        if (lp_info->A->value) free(lp_info->A->value);
        free(lp_info->A);
        lp_info->A = nullptr;
    }
    
    // Free constraint and variable bound vectors
    if (lp_info->AL) {
        free(lp_info->AL);
        lp_info->AL = nullptr;
    }
    if (lp_info->AU) {
        free(lp_info->AU);
        lp_info->AU = nullptr;
    }
    if (lp_info->l) {
        free(lp_info->l);
        lp_info->l = nullptr;
    }
    if (lp_info->u) {
        free(lp_info->u);
        lp_info->u = nullptr;
    }
    if (lp_info->c) {
        free(lp_info->c);
        lp_info->c = nullptr;
    }
}